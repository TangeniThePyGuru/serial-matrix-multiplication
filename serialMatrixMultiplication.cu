#include <stdio.h>
#include <assert.h>
#include "support.h"

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


// void initWith(float num, float *a, int N)
// {
//   for(int i = 0; i < N; ++i)
//   {
//     a[i] = num;
//   }
// }

void multMatricesInto(float *result, float *a, float *b, int N)
{

  for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			result[i][j] = 0.0;
			for (k = 0; k < N; k++)
				result[i][j] = c[i][j] + a[i][k] * b[k][j];
		} /* end j loop */
	}
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values multiplied correctly.\n");
}

int main(int argc, char **argv)
{
//   const int N = 2<<20;
//   size_t size = N * sizeof(float);

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
//   size_t A_sz, B_sz, C_sz;
  
  Timer timer;

  // get array sizes

  if (argc == 1) {
      numARows = 1000;
      numAColumns = numBRows = 1000;
      numBColumns = 1000;
  } else if (argc == 2) {
      numARows = atoi(argv[1]);
      numAColumns = numBRows = atoi(argv[1]);
      numBColumns = atoi(argv[1]);
  } else if (argc == 4) {
      numARows = atoi(argv[1]);
      numAColumns = numBRows = atoi(argv[2]);
      numBColumns = atoi(argv[3]);
  } else {
      printf("\n    Invalid input parameters!"
          "\n    Usage: ./lab3                # All matrices are 1000 x 1000"
          "\n    Usage: ./lab3 <m>            # All matrices are m x m"
          "\n    Usage: ./lab3 <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
          "\n");
      exit(0);
  }

  // set the C matrix row and col size
  numCRows = numARows;
  numCColumns = numBColumns;

  float a[numARows][numAColumns];
  float b[numARows][numBColumns];
  float c[numCRows][numCColumns];

  // matrix sizes
//   A_sz = numARows * numAColumns;
//   B_sz = numBRows * numBColumns;
//   C_sz = numCRows * numCColumns;

  //@@ Allocate CPU memory and assign data

//   a = (float*) malloc( sizeof(float)*A_sz );
  for (unsigned int i=0; j < numARows; j++) { 
    for (unsigned int j=0; i < numAColumns; i++)  {
        a[j][i] = (rand()%100)/100.00; 
    }
  }


//   b = (float*) malloc( sizeof(float)*B_sz );
  for (unsigned int j=0; j < numBRows; j++) {
    for (unsigned int i=0; i < numBColumns; i++) { 
        b[j][i] = (rand()%100)/100.00; 
    }
  }

//   c = (float*) malloc( sizeof(float)*C_sz );

  printf("Performing Mutrix Multiplication..."); fflush(stdout);
  startTime(&timer);

  multMatricesInto(c, a, b, numCRows);


  printf("Verifying Mutrix Multiplication..."); fflush(stdout);
  startTime(&timer);
  // verify
//   checkElementsAre(7, c, C_sz);

//   free(a);
//   free(b);
//   free(c);
}
    